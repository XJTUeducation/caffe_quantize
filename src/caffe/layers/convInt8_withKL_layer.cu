#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include "caffe/layers/convInt8_withKL_layer.hpp"
//#include "caffe/util/im2col.hpp"
namespace caffe {

using std::cout;
using std::endl;


int getNewDim(int n, int k,int*newN,int*newK)
{
	if(n%4==0 && k%4==0)
	{
		*newN = n;
		*newK = k;
		return 0;
	}	
	*newN = n%4==0? n : (n/4+1)*4;
	*newK = k%4==0? k : (k/4+1)*4;
	return 1;
}


template <typename Dtype>
void ConvInt8withKLLayer<Dtype>::computeInt8Weight(int *idx,const Dtype t1,const Dtype t2)
{
  Dtype t1_pos = t1>0?t1:(-t1);
  //Dtype t2_pos = t2>0?t2:(-t2);
  Dtype each_seg = (t2-t1);
  int this_idx = (*idx-this->preTestBatches);
  int whichWeightSeg=this_idx%(weight_adjust_segment_count*weight_adjust_each_count)/weight_adjust_each_count;
  if(current_weight_adjust_segment_idx != whichWeightSeg)
  {
    current_weight_adjust_segment_idx = whichWeightSeg;
    Dtype this_t = t1_pos+each_seg*whichWeightSeg/input_adjust_segment_count;
    this->blobs_[0].get()->mutable_cpu_data()[4]=this_t*-1;
    this->blobs_[0].get()->mutable_cpu_data()[5]=this_t;
    this->weight_temp_unit_sacle = this_t/127;
    this->weight_temp_unit_sacle_1 = 127.0/this_t;
#ifdef SHOW_WEIGHT2INT8
    std::cout<<"--------weight_T : "<<this_t<<"--------"<<std::endl;
    std::cout<<"--------before 2int8--------"<<std::endl;
    showDevice(this->blobs_int8_[0].get()->gpu_data(),20);
#endif
    weight2int8(weightFp32.count(),weightFp32.gpu_data(),this->blobs_int8_[0].get()->mutable_gpu_data(),this_t*-1.0,this_t,weight_temp_unit_sacle_1,0,false);
#ifdef SHOW_WEIGHT2INT8
    std::cout<<"--------after 2int8--------"<<std::endl;
    showDevice(this->blobs_int8_[0].get()->gpu_data(),20);
    std::cout<<"--------weightFp32--------"<<std::endl;
    showDevice(weightFp32.gpu_data(),20);
#endif
  }
  LOG(INFO)<<"weight_temp_unit_sacle = "<<this->weight_temp_unit_sacle<<";  weight_temp_unit_sacle_1 = "<<this->weight_temp_unit_sacle_1;
}
template <typename Dtype>
void ConvInt8withKLLayer<Dtype>::computeInt8input(int *idx, const Dtype t1,const Dtype t2)
{
  Dtype t1_pos = t1>0?t1:(-t1);
  //Dtype t2_pos = t2>0?t2:(-t2);
  Dtype each_seg = (t2-t1);
  int this_idx = (*idx-this->preTestBatches);
  int whichInputSeg=this_idx/(weight_adjust_segment_count*weight_adjust_each_count)/input_adjust_each_count;
  Dtype this_t = t1_pos+each_seg*whichInputSeg/input_adjust_segment_count;
  this->blobs_[0].get()->mutable_cpu_data()[2]=this_t*-1;
  this->blobs_[0].get()->mutable_cpu_data()[3]=this_t;
  this->input_temp_unit_sacle = this_t/127;
  this->input_temp_unit_sacle_1 = 127.0/this_t;

  LOG(INFO)<<"input_temp_unit_sacle = "<<this->input_temp_unit_sacle<<";  input_temp_unit_sacle_1 = "<<this->input_temp_unit_sacle_1;
}

template <typename Dtype>
void ConvInt8withKLLayer<Dtype>::weight2int8(const int count, const Dtype*fp32weights, signed char*int8weight, const Dtype minT, const Dtype maxT, const Dtype unit_scale, const Dtype bias, bool doBias)
{
  Dtype weight_uni_scale=0;
  if(unit_scale>0)
  {weight_uni_scale=unit_scale;}
  else
  {weight_uni_scale=254.0/(maxT-minT);}
  if(!doBias)
  {
    CHECK(minT < maxT);
    caffe_gpu_quantize_nobias(count, fp32weights, int8weight, minT, maxT, weight_uni_scale);//这里是希望做乘法
  }
  else
  {
    LOG(FATAL)<<"NOT IMPELMENT!";
  }
}


template <typename Dtype>
void ConvInt8withKLLayer<Dtype>::forward_gpu_gemm(const Dtype* input,
    const signed char* weights, Dtype* output, bool skip_im2col) {
#ifdef SHOW_INPUT2INT8

    std::cout<<"-------------input----------------"<<std::endl; 
    showDevice(input,50); 
    std::cout<<"----------------------before col_buffer_----------------------"<<std::endl; 
    showDevice(col_buffer_.cpu_data(),50); 
#endif
#ifdef SHOW_FP32COL    
   if (!is_1x1_) 
   {
     std::cout<<"-------------input SHOW_INPUT2INT8-------------"<<std::endl; 
          showDevice(input,50); 
          im2col_gpu(input, conv_in_channels_,
                      conv_input_shape_.cpu_data()[1], conv_input_shape_.cpu_data()[2],
                      kernel_shape_.cpu_data()[0], kernel_shape_.cpu_data()[1],
                      pad_.cpu_data()[0], pad_.cpu_data()[1],
                      stride_.cpu_data()[0], stride_.cpu_data()[1],
                      dilation_.cpu_data()[0], dilation_.cpu_data()[1], col_buffer_show_.mutable_gpu_data());

        std::cout<<"--------------fp32 col_buffer_show_---------------"<<std::endl; 
        showDevice(col_buffer_show_.gpu_data(),500); 
   }
#endif
  const signed char* col_buff;
  if (!is_1x1_) {

    
          CHECK( num_spatial_axes_ == 2);
          im2col_gpu_quantized(input, conv_in_channels_, conv_input_shape_.cpu_data()[1], conv_input_shape_.cpu_data()[2],
                                                    kernel_shape_.cpu_data()[0], kernel_shape_.cpu_data()[1],
                                                    pad_.cpu_data()[0], pad_.cpu_data()[1],
                                                    stride_.cpu_data()[0], stride_.cpu_data()[1],
                                                    dilation_.cpu_data()[0], dilation_.cpu_data()[1], col_buffer_.mutable_gpu_data(),this->blobs_[0].get()->cpu_data()[2],this->blobs_[0].get()->cpu_data()[3],this->input_temp_unit_sacle_1);
    col_buff = col_buffer_.gpu_data();
#ifdef SHOW_INPUT2INT8
          std::cout<<"--------------input_T1 = "<<this->blobs_[0].get()->cpu_data()[2]<<"-------------"<<std::endl; 
          std::cout<<"--------------input_T2 = "<<this->blobs_[0].get()->cpu_data()[3]<<"-------------"<<std::endl; 
          std::cout<<"--------------input_sacle = "<<this->input_temp_unit_sacle_1<<"-------------"<<std::endl; 
          std::cout<<"-------------after col_buffer_-------------"<<std::endl; 
          showDevice(col_buffer_.gpu_data(),500);
          
#endif
  }
  else
  {
        im2col_1x1_gpu_quantized(col_buffer_.count(), input, col_buffer_.mutable_gpu_data(), this->blobs_[0].get()->mutable_cpu_data()[2],this->blobs_[0].get()->mutable_cpu_data()[3],this->input_temp_unit_sacle);
        col_buff = col_buffer_.gpu_data();
  }
  	int newK=0;
	int newN=0;
	signed char *d_A_new, *d_B_new;
	int * d_C_32_new;
	int m = conv_out_channels_ / group_;
	int needReshape = getNewDim(conv_out_spatial_dim_,kernel_dim_,&newN,&newK);
	int bigger_count1=m*newK;
	int bigger_count2=newK*newN;
	int bigger_count=m*newN;
	LOG(INFO)<<"needReshape = "<<needReshape<<" AND (newN,newK)=( "<<newN<<", "<<newK<<" ) THE OLD = ("<<conv_out_spatial_dim_<<", "<<kernel_dim_<<" )";
	if(needReshape>0) 
	{
		(hipMalloc(&d_A_new, m * newK * sizeof(signed char)));
		(hipMalloc(&d_B_new, newK * newN * sizeof(signed char)));
		(hipMalloc(&d_C_32_new, m * newN * sizeof(signed char)));
	}
	
  for (int g = 0; g < group_; ++g) 
  {
	if(needReshape>0) 
	{  
	  	_copy_Data<<<CAFFE_GET_BLOCKS(bigger_count1), CAFFE_CUDA_NUM_THREADS>>>(bigger_count1,weights + weight_offset_ * g,d_A_new,m,kernel_dim_,m,newK);
		_copy_Data<<<CAFFE_GET_BLOCKS(bigger_count2), CAFFE_CUDA_NUM_THREADS>>>(bigger_count2,col_buff + col_offset_ * g,d_B_new,kernel_dim_,conv_out_spatial_dim_,newK,newN);
		caffe_gpu_iGemm(CblasNoTrans, CblasNoTrans, m, newN, newK,
        1, d_A_new, d_B_new,
        0, d_C_32_new);
		_copy_Data_back<<<CAFFE_GET_BLOCKS(bigger_count), CAFFE_CUDA_NUM_THREADS>>>(bigger_count,int32out.mutable_gpu_data() + output_offset_ * g,d_C_32_new,m,conv_out_spatial_dim_,m,newN);
	}  
	else
	{
		caffe_gpu_iGemm(CblasNoTrans, CblasNoTrans, m, newN, newK,
        1, weights + weight_offset_ * g, col_buff + col_offset_ * g,
        0, int32out.mutable_gpu_data() + output_offset_ * g);
	}
  }
  if(needReshape>0) 
  {  
	hipFree(d_A_new);
	hipFree(d_B_new);
	hipFree(d_C_32_new);
  }

    
  int2Dtype(int32out.count(),int32out.gpu_data(),output,this->input_temp_unit_sacle*this->weight_temp_unit_sacle);

}
template <typename Dtype>
void ConvInt8withKLLayer<Dtype>::forward_gpu_bias(Dtype* output, const Dtype* bias)
{}


__global__ void getRelativeEntropy(int N, const float*data, const float*gtData,float*outData, float relativeLog)
{
	CUDA_KERNEL_LOOP(idx,N){
		//outData[idx] = 1.0;//gtData[idx]*(log(data[idx]/gtData[idx])-relativeLog);
		outData[idx] = gtData[idx]*(log(data[idx]/gtData[idx])-relativeLog);
	}
}
__global__ void getRelativeEntropy(int N, const double*data, const double*gtData,double*outData, double relativeLog)
{
	CUDA_KERNEL_LOOP(idx,N){
		outData[idx] = gtData[idx]*(log(data[idx]/gtData[idx])-relativeLog);
	}
}

template <typename Dtype>
Dtype ConvInt8withKLLayer<Dtype>::figureRelativeEntropy(Blob<Dtype>*int8Blob, Blob<Dtype>*gtBlob)
{
	
	CHECK(int8Blob->count() == gtBlob->count());
	CHECK(int8Blob->count() == this->relativeEntropyBlob.count());
	
	caffe_gpu_abs(int8Blob->count(), int8Blob->gpu_data(), int8Blob->mutable_gpu_data());
	caffe_gpu_abs(int8Blob->count(), gtBlob->gpu_data(), gtBlob->mutable_gpu_data());
	LOG(INFO)<<int8Blob->asum_data();//有同步问题！！！！
	Dtype int8Sum = int8Blob->asum_data();
	Dtype gtSum = gtBlob->asum_data();
	Dtype relativeLog = log(gtSum/int8Sum);
	LOG(INFO)<<int8Sum;
	//return 0;
	int _count=int8Blob->count();
	getRelativeEntropy<<<CAFFE_GET_BLOCKS(_count), CAFFE_CUDA_NUM_THREADS>>>(
      int8Blob->count(), int8Blob->gpu_data(), gtBlob->gpu_data(), this->relativeEntropyBlob.mutable_gpu_data(),relativeLog);

	Dtype entropy_=-1;
	LOG(INFO)<<int8Blob->count();
	LOG(INFO)<<this->relativeEntropyBlob.count();
	LOG(INFO)<<relativeLog;
    //caffe_gpu_asum(relativeEntropyBlob.count(), relativeEntropyBlob.gpu_data(), &(relativeEntropyBlob.mutable_gpu_diff()[0]));
	
	//showDevice(this->relativeEntropyBlob.gpu_data(),50);
	//showDevice(this->relativeEntropyBlob.gpu_data(),int8Blob->count());
	
	LOG(INFO)<<this->relativeEntropyBlob.asum_data();
	exit(0);
	//LOG(INFO)<<gtSum;
	//LOG(INFO)<<relativeLog;
	//std::cout<<entropy_<<std::endl;
	return entropy_/gtSum;
}


template <typename Dtype>
void ConvInt8withKLLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
 
    if(!weightFp32HasExtracted)  
    {
        getFp32Weight();
        getMaxAndMIn(weightFp32.count(),weightFp32.gpu_data(), &(this->maxAndMin.mutable_gpu_data()[3]),&(this->maxAndMin.mutable_gpu_data()[2]));
        LOG(INFO)<<this->maxAndMin.cpu_data()[3]<<"\t\t"<<this->maxAndMin.cpu_data()[2];
    }

    if(preTestIdx<preTestBatches)
    {
        getMaxAndMIn(bottom[0]->count(),bottom[0]->gpu_data(), &(this->maxAndMin.mutable_gpu_data()[1]),&(this->maxAndMin.mutable_gpu_data()[0]));
        preTestIdx++;
        if(!isFirstGetMaxMin)
        {
            input_scale_t1=this->maxAndMin.cpu_data()[0];
            input_scale_t2=this->maxAndMin.cpu_data()[1];
            isFirstGetMaxMin = true;
        }
        else
        {
          this->maxAndMin.mutable_cpu_data()[0] = this->maxAndMin.cpu_data()[0]>input_scale_t1?input_scale_t1:this->maxAndMin.cpu_data()[0];
          this->maxAndMin.mutable_cpu_data()[1] = this->maxAndMin.cpu_data()[1]<input_scale_t2?input_scale_t2:this->maxAndMin.cpu_data()[1];
          input_scale_t1=this->maxAndMin.cpu_data()[0];
          input_scale_t2=this->maxAndMin.cpu_data()[1];
        }
        LOG(INFO)<<"input  region : "<<this->maxAndMin.cpu_data()[1]<<"\t"<<this->maxAndMin.cpu_data()[0];
        LOG(INFO)<<"weight region : "<<this->maxAndMin.cpu_data()[3]<<"\t"<<this->maxAndMin.cpu_data()[2];
        LOG(INFO)<<preTestIdx<<" < "<<preTestBatches;
    }

    if(preTestIdx<preTestBatches) return;
    computeInt8Weight(&preTestIdx,this->maxAndMin.cpu_data()[2],this->maxAndMin.cpu_data()[3]);
    computeInt8input(&preTestIdx,this->maxAndMin.cpu_data()[0],this->maxAndMin.cpu_data()[1]);
    preTestIdx++;

  const signed char* weight = this->blobs_int8_[0]->gpu_data();
  //for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top_result.mutable_gpu_data();
    for (int n = 0; n < this->num_; ++n) {
      this->forward_gpu_gemm(bottom_data + n * this->bottom_dim_, weight,
          top_data + n * this->top_dim_);
		  LOG(INFO)<<"forward_gpu_gemm DONE";

      if (this->bias_term_) {
		  LOG(INFO)<<"bias_term_ BEGIN";
		  LOG(INFO)<<"this->blobs_[1]->SHAPE_STRING = "<<this->blobs_[1]->shape_string();
		  LOG(INFO)<<top_result.shape_string();
		  LOG(INFO)<<this->blobs_[1]->cpu_data()[1];
        const Dtype* bias = this->blobs_[1]->gpu_data();
          caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_output_,
                  out_spatial_dim_, 1, (Dtype)1., bias, bias_multiplier_.gpu_data(),
                  (Dtype)1., top_data + n * this->top_dim_);
		LOG(INFO)<<"bias_term_ DONE";
      }
    }

    // std::cout<<"*************int8 result*************"<<std::endl; 
    // showDevice(top_result.cpu_data(),50);
    // std::cout<<"*************fp32 result*************"<<std::endl; 
    // showDevice(bottom[1]->cpu_data(),50);
    // std::cout<<"============================"<<std::endl; 
	//LOG(INFO)<<top_result.asum_data();
	//LOG(INFO)<<bottom[1]->asum_data();
	//figureRelativeEntropy(&top_result, bottom[1]);
	
	top[0]->mutable_cpu_data()[0]=figureRelativeEntropy(&top_result, bottom[1]);
}

template <typename Dtype>
void ConvInt8withKLLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
        LOG(INFO)<<2;
    LOG(FATAL)<<"NOT IMPLEMENTED";
}

INSTANTIATE_LAYER_GPU_FUNCS(ConvInt8withKLLayer);

}  // namespace caffe
