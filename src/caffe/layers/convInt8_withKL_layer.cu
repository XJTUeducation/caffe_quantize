#include <vector>
#include <iostream>
#include "caffe/layers/convInt8_withKL_layer.hpp"
//#include "caffe/util/im2col.hpp"
namespace caffe {

using std::cout;
using std::endl;

template <typename Dtype>
void showDevice3(const Dtype*data,int count)
{
    Dtype *show=(Dtype*)malloc(count*sizeof(Dtype));
    hipMemcpy(show,data,count*sizeof(Dtype),hipMemcpyDeviceToHost);
    for(int i=0;i<count;i++)
    {
        std::cout<<(float)show[i]<<" ";
        if(i%10==9)std::cout <<std::endl;
    }
    free(show);
}

template <typename Dtype>
void ConvInt8withKLLayer<Dtype>::computeInt8Weight(int *idx,const Dtype t1,const Dtype t2)
{
  Dtype t1_pos = t1>0?t1:(-t1);
  Dtype t2_pos = t2>0?t2:(-t2);
  Dtype each_seg = (t2-t1);
  int this_idx = (*idx-this->preTestBatches);
  int whichWeightSeg=this_idx%(weight_adjust_segment_count*weight_adjust_each_count)/weight_adjust_each_count;
  if(current_weight_adjust_segment_idx != whichWeightSeg)
  {
    current_weight_adjust_segment_idx = whichWeightSeg;
    Dtype this_t = t1_pos+each_seg*whichWeightSeg/input_adjust_segment_count;
    this->blobs_[0].get()->mutable_cpu_data()[4]=this_t;
    this->blobs_[0].get()->mutable_cpu_data()[5]=this_t;
    this->weight_temp_unit_sacle = this_t/127;
    this->weight_temp_unit_sacle_1 = 127.0/this_t;
    
    TODO:
  }
  LOG(INFO)<<"weight_temp_unit_sacle = "<<this->weight_temp_unit_sacle<<";  weight_temp_unit_sacle_1 = "<<this->weight_temp_unit_sacle_1;
}
template <typename Dtype>
void ConvInt8withKLLayer<Dtype>::computeInt8input(int *idx,const Dtype t1,const Dtype t2)
{
  Dtype t1_pos = t1>0?t1:(-t1);
  Dtype t2_pos = t2>0?t2:(-t2);
  Dtype each_seg = (t2-t1);
  int this_idx = (*idx-this->preTestBatches);
  int whichInputSeg=this_idx/(weight_adjust_segment_count*weight_adjust_each_count)/input_adjust_each_count;
  Dtype this_t = t1_pos+each_seg*whichInputSeg/input_adjust_segment_count;
  this->blobs_[0].get()->mutable_cpu_data()[2]=this_t;
  this->blobs_[0].get()->mutable_cpu_data()[3]=this_t;

  this->input_temp_unit_sacle = this_t/127;
  this->input_temp_unit_sacle_1 = 127.0/this_t;

  LOG(INFO)<<"input_temp_unit_sacle = "<<this->input_temp_unit_sacle<<";  input_temp_unit_sacle_1 = "<<this->input_temp_unit_sacle_1;
}

template <typename Dtype>
void ConvInt8withKLLayer<Dtype>::weight2int8(const int count, const Dtype*fp32weights, signed char*int8weight, const Dtype minT, const Dtype maxT, const Dtype unit_scale, const Dtype bias, bool doBias)
{
  Dtype weight_uni_scale=0;
  if(unit_scale>0)
  {weight_uni_scale=unit_scale;}
  else
  {weight_uni_scale=(maxT-minT)/255;}
  if(!doBias)
  {
    CHECK(minT < maxT);
    caffe_gpu_quantize_nobias(count, fp32weights, int8weight, minT, maxT, unit_scale);
  }
  else
  {
    LOG(FATAL)<<"NOT IMPELMENT!";
  }
}


template <typename Dtype>
void ConvInt8withKLLayer<Dtype>::forward_gpu_gemm(const Dtype* input,
    const signed char* weights, Dtype* output, bool skip_im2col) {
      
  const signed char* col_buff;

  if (!is_1x1_) {
          CHECK( num_spatial_axes_ == 2);
          // std::cout<<"----------------------input----------------------"<<std::endl; // showDevice3(input,10); // std::cout<<"----------------------initi  col_buffer_----------------------"<<std::endl; // showDevice3(col_buffer_.gpu_data(),50); // printf("-----   %p\n",col_buffer_.mutable_gpu_data()); // printf("-----   %p\n",col_buffer_.gpu_data());
          im2col_gpu_quantized(input, conv_in_channels_, conv_input_shape_.cpu_data()[1], conv_input_shape_.cpu_data()[2],
                                                    kernel_shape_.cpu_data()[0], kernel_shape_.cpu_data()[1],
                                                    pad_.cpu_data()[0], pad_.cpu_data()[1],
                                                    stride_.cpu_data()[0], stride_.cpu_data()[1],
                                                    dilation_.cpu_data()[0], dilation_.cpu_data()[1], col_buffer_.mutable_gpu_data(),(Dtype)-0.4,(Dtype)0.4,(Dtype)317.5);
    col_buff = col_buffer_.gpu_data();
  }
  else
  {
        im2col_1x1_gpu_quantized(col_buffer_.count(), input, col_buffer_.mutable_gpu_data(), (Dtype)-0.4, (Dtype)0.4, (Dtype)317.5);
        col_buff = col_buffer_.gpu_data();
  }

  for (int g = 0; g < group_; ++g) {
    caffe_gpu_iGemm(CblasNoTrans, CblasNoTrans, conv_out_channels_ /
        group_, conv_out_spatial_dim_, kernel_dim_,
        1, weights + weight_offset_ * g, col_buff + col_offset_ * g,
        0, int32out.mutable_gpu_data() + output_offset_ * g);
  }

showDevice3(int32out.gpu_data(),50);
  int2Dtype(int32out.count(),int32out.gpu_data(),output,(Dtype)0.001);
  std::cout<<"----------------------output----------------------"<<std::endl; 
  showDevice3(output,50); 
  
#ifdef SHOW_FP32_OUT
   if (!is_1x1_) 
   {
        im2col_gpu(input, conv_in_channels_, conv_input_shape_.cpu_data()[1], conv_input_shape_.cpu_data()[2],
                                                  kernel_shape_.cpu_data()[0], kernel_shape_.cpu_data()[1],
                                                  pad_.cpu_data()[0], pad_.cpu_data()[1],
                                                  stride_.cpu_data()[0], stride_.cpu_data()[1],
                                                  dilation_.cpu_data()[0], dilation_.cpu_data()[1], col_buffer_show_.mutable_gpu_data());
   }
#endif

}
template <typename Dtype>
void ConvInt8withKLLayer<Dtype>::forward_gpu_bias(Dtype* output, const Dtype* bias)
{}


template <typename Dtype>
void ConvInt8withKLLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
    if(!weightFp32HasExtracted)  
    {
        getFp32Weight();
        getMaxAndMIn(weightFp32.count(),weightFp32.gpu_data(), &(this->maxAndMin.mutable_gpu_data()[3]),&(this->maxAndMin.mutable_gpu_data()[2]));
        LOG(INFO)<<this->maxAndMin.cpu_data()[3]<<"\t\t"<<this->maxAndMin.cpu_data()[2];
    }
    if(preTestIdx<preTestBatches)
    {
        getMaxAndMIn(bottom[0]->count(),bottom[0]->gpu_data(), &(this->maxAndMin.mutable_gpu_data()[1]),&(this->maxAndMin.mutable_gpu_data()[0]));
        preTestIdx++;
        if(!isFirstGetMaxMin)
        {
            input_scale_t1=this->maxAndMin.cpu_data()[0];
            input_scale_t2=this->maxAndMin.cpu_data()[1];
            isFirstGetMaxMin = true;
        }
        else
        {
          this->maxAndMin.mutable_cpu_data()[0] = this->maxAndMin.cpu_data()[0]>input_scale_t1?input_scale_t1:this->maxAndMin.cpu_data()[0];
          this->maxAndMin.mutable_cpu_data()[1] = this->maxAndMin.cpu_data()[1]<input_scale_t2?input_scale_t2:this->maxAndMin.cpu_data()[1];
          input_scale_t1=this->maxAndMin.cpu_data()[0];
          input_scale_t2=this->maxAndMin.cpu_data()[1];
        }
        LOG(INFO)<<"input  region : "<<this->maxAndMin.cpu_data()[1]<<"\t\t"<<this->maxAndMin.cpu_data()[0];
        LOG(INFO)<<"weight region : "<<this->maxAndMin.cpu_data()[3]<<"\t\t"<<this->maxAndMin.cpu_data()[2];
        LOG(INFO)<<preTestIdx<<" < "<<preTestBatches;
    }
    if(preTestIdx<preTestBatches) return;
    computeInt8Weight(&preTestIdx,this->maxAndMin.cpu_data()[2],this->maxAndMin.cpu_data()[3]);
    computeInt8input(&preTestIdx,this->maxAndMin.cpu_data()[0],this->maxAndMin.cpu_data()[1]);
    preTestIdx++;
    return;
  const signed char* weight = this->blobs_int8_[0]->gpu_data();
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();
    for (int n = 0; n < this->num_; ++n) {
      this->forward_gpu_gemm(bottom_data + n * this->bottom_dim_, weight,
          top_data + n * this->top_dim_);
      if (this->bias_term_) {
        const Dtype* bias = this->blobs_[1]->gpu_data();
        this->forward_gpu_bias(top_data + n * this->top_dim_, bias);
      }
    }
  }
}

template <typename Dtype>
void ConvInt8withKLLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
        LOG(INFO)<<2;
    LOG(FATAL)<<"NOT IMPLEMENTED";
}

INSTANTIATE_LAYER_GPU_FUNCS(ConvInt8withKLLayer);

}  // namespace caffe
